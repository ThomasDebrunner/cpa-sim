
#include <hip/hip_runtime.h>
__global__ void bgr_to_gray_kernel(unsigned char* input,
                     unsigned char* output,
                     int width,
                     int height,
                     int colorWidthStep,
                     int grayWidthStep)
{
    //2D Index of current thread
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    //Only valid threads perform memory I/O
    if((xIndex < width) && (yIndex <  height))
    {
        /* Kernel Code Here */
    }
}